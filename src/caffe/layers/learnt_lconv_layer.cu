#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layers/learnt_lconv_layer.hpp"
#include "caffe/util/math_functions.hpp"



namespace caffe {

template <typename Dtype>
__global__ void LearntLConvForward(const int nthreads,
          const Dtype* bottom_data, const int num, const int channels,
		  const int height, const int width, const Dtype* filter, const int filter_height,
    	  const int filter_width, Dtype* top_data) 
{
	  CUDA_KERNEL_LOOP(index, nthreads) {
	  		int w = index % width;
	  		int h = (index / width) % height;
	  		int c = (index / width / height) % channels;
	  		int n = index / width / height / channels;
	  		for (int i=0; i<filter_height; i++)
	  		{
	  			int n_h = h + i  - (filter_height / 2);
	  			if (n_h >= 0 and n_h < height)
		  			for (int j=0; j<filter_width; j++)
		  			{
		  				int n_w = w + j  - (filter_width / 2);
		  				if (n_w >= 0 and n_w < width)
		  					top_data[index] += bottom_data[((n * channels  + c) * height + n_h)*width + n_w] * filter[((n * filter_height * filter_width  + i*filter_width + j) * height + h)*width + w];
	  				}
  			}
	  }
}


template <typename Dtype>
__global__ void LearntLConvBackward(const int nthreads,
          Dtype* bottom_diff, const Dtype* bottom_data, const int num, const int channels,
		  const int height, const int width, Dtype* filter_diff, const Dtype* filter, const int filter_height,
    	  const int filter_width, const Dtype* top_diff) 
{
	  CUDA_KERNEL_LOOP(index, nthreads) {
	  		int w = index % width;
	  		int h = (index / width) % height;
	  		int c = (index / width / height) % channels;
	  		int n = index / width / height / channels;
	  		for (int i=0; i<filter_height; i++)
	  		{
	  			int n_h = h + i  - (filter_height / 2);
	  			if (n_h >= 0 and n_h < height)
		  			for (int j=0; j<filter_width; j++)
		  			{
		  				int n_w = w + j  - (filter_width / 2);
		  				if (n_w >= 0 and n_w < width)
		  				{
		  					bottom_diff[((n * channels  + c) * height + n_h)*width + n_w] += top_diff[index] * filter[((n * filter_height * filter_width  + i*filter_width + j) * height + h)*width + w];
		  					filter_diff[((n * filter_height * filter_width  + i*filter_width + j) * height + h)*width + w] += top_diff[index] * bottom_data[((n * channels  + c) * height + n_h)*width + n_w];
  						}
  					}
			}
	  }
}

template <typename Dtype>
void LearntLConvLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {

  const Dtype* bottom_data = bottom[0]->gpu_data();
  const Dtype* bottom_filter = bottom[1]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  int count = top[0]->count();
  caffe_gpu_set(count, Dtype(0.), top_data);

  int filter_height = int(std::sqrt(bottom[1]->channels()));
  int filter_width = filter_height;
  CHECK_EQ(filter_height * filter_width, bottom[1]->channels());

  LearntLConvForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, bottom_data, top[0]->num(), top[0]->channels(), top[0]->height(), top[0]->width(), bottom_filter, filter_height, filter_width, top_data);
  
  CUDA_POST_KERNEL_CHECK;
}

template <typename Dtype>
void LearntLConvLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {

  int count = top[0]->count();
  caffe_gpu_set(bottom[0]->count(), Dtype(0.), bottom[0]->mutable_gpu_diff());
  caffe_gpu_set(bottom[1]->count(), Dtype(0.), bottom[1]->mutable_gpu_diff());

  int filter_height = int(std::sqrt(bottom[1]->channels()));
  int filter_width = filter_height;
  CHECK_EQ(filter_height * filter_width, bottom[1]->channels());

  LearntLConvBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, bottom[0]->mutable_gpu_diff(), bottom[0]->gpu_data(), top[0]->num(), top[0]->channels(), top[0]->height(), top[0]->width(), bottom[1]->mutable_gpu_diff(), bottom[1]->gpu_data(), filter_height, filter_width, top[0]->gpu_diff());
  
  CUDA_POST_KERNEL_CHECK;
}


INSTANTIATE_LAYER_GPU_FUNCS(LearntLConvLayer);


}  // namespace caffe