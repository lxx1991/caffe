#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/layers/feature_selection_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
__global__ void FeatureSelectionForward(const int nthreads,
    const Dtype* const bottom_data, const Dtype* const bottom_index, const int channels,
    const int height, const int height_index, const int width_index, 
    Dtype* const top_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int w = index % width_index;
    const int c = (index / width_index) % channels;
    const int n = index / width_index / channels;
    const int n_idx = int(bottom_index[n * width_index + w]);
    top_data[index] = bottom_data[c *  height + n_idx];
  }
}



template <typename Dtype>
void FeatureSelectionLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  const Dtype* bottom_index = bottom[1]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  int count = top[0]->count();

  FeatureSelectionForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, bottom_data, bottom_index, bottom[0]->channels(),
        bottom[0]->height(), bottom[1]->height(), bottom[1]->width(), top_data);

  CUDA_POST_KERNEL_CHECK;
}


template <typename Dtype>
__global__ void FeatureSelectionBackward(const int nthreads, const Dtype* const top_diff, const int channels,
    const int height, const int width,
    const int* const ending, const int* const prev, const int* const head,
    Dtype* const bottom_diff) {

  CUDA_KERNEL_LOOP(index, nthreads) {
    const int h = index % height;
    const int c = index / height;
    for (int n = head[h]; n!=-1; n = prev[n])
      bottom_diff[index] += top_diff[ (ending[n] / width) * channels * width + c * width + (ending[n] % width) ];
  }
}

template <typename Dtype>
void FeatureSelectionLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  if (!propagate_down[0]) {
    return;
  }
  const Dtype* top_diff = top[0]->gpu_diff();
  const Dtype* bottom_index = bottom[1]->cpu_data();
  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
  int *head = head_.mutable_cpu_data(), *prev = prev_.mutable_cpu_data(), *ending = ending_.mutable_cpu_data();
  caffe_set(bottom[0]->height(), -1, head);
  int index = 0;
  for (int h=0; h<bottom[1]->height(); h++)
    for (int w=0; w<bottom[1]->width(); w++)
    {
      int x = int(bottom_index[index]), y = h * bottom[1]->width() + w;
      ending[index] = y;
      prev[index] = head[x];
      head[x] = index;
      index = index + 1;
    }
  const int count = bottom[0]->count();
  caffe_gpu_set(count, Dtype(0.), bottom_diff);
  //LOG(ERROR) << "start";
  FeatureSelectionBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, top_diff, bottom[0]->channels(), bottom[0]->height(), bottom[1]->width(), 
        ending_.gpu_data(), prev_.gpu_data(), head_.gpu_data(), bottom_diff);
  //LOG(ERROR) << "end";
  CUDA_POST_KERNEL_CHECK;
}


INSTANTIATE_LAYER_GPU_FUNCS(FeatureSelectionLayer);


}  // namespace caffe
