#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>
#include <map>


#include "caffe/layers/superpixel_pooling_layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/filler.hpp"


namespace caffe {

using std::map;

template <typename Dtype>
__global__ void AveSuperpixelPoolForward(const int nthreads,
    const Dtype* const bottom_data, const int channels,
    const int height, const int width, const int superpixel_num_,
    const int* head, const int* ending, const Dtype* weight, const int* prev,
    Dtype* const top_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int h_t = index % superpixel_num_;
    const int c = (index / superpixel_num_) % channels;
    const int base_index = c * height * width;
    for (int h=head[h_t]; h!=-1; h = prev[h])
      top_data[index] += bottom_data[base_index + ending[h]] * weight[h];
  }
}


template <typename Dtype>
void Fill(Blob<Dtype>* blob) {
    CHECK_EQ(blob->num_axes(), 4) << "Blob must be 4 dim.";
    CHECK_EQ(blob->width(), blob->height()) << "Filter must be square";
    Dtype* data = blob->mutable_cpu_data();
    int f = ceil(blob->width() / 2.);
    float c = (2 * f - 1 - f % 2) / (2. * f);
    for (int i = 0; i < blob->count(); ++i) {
      float x = i % blob->width();
      float y = (i / blob->width()) % blob->height();
      data[i] = (1 - fabs(x / f - c)) * (1 - fabs(y / f - c));
    }
 }


template <typename Dtype>
void SuperpixelPoolingLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {

  if (this->layer_param_.pooling_param().pool() != PoolingParameter_PoolMethod_AVE)
  {
    NOT_IMPLEMENTED;
    return;
  }

  const Dtype* bottom_sp = bottom[1]->cpu_data();
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();

  int count = top[0]->count();

  int kernel_size = 2 * factor_ - factor_ % 2, pad = ceil((factor_ - 1) / 2.);
  Blob<Dtype> filter(1, 1, kernel_size, kernel_size);
  Fill(&filter);
  int h = bottom[0]->height(), w = bottom[0]->width(), sh = bottom[1]->height(), sw = bottom[1]->width();

  int *ending = ending_.mutable_cpu_data(), *prev = prev_.mutable_cpu_data(), *head = head_.mutable_cpu_data();
  int *ending_bp = ending_bp_.mutable_cpu_data(), *prev_bp = prev_bp_.mutable_cpu_data(), *head_bp = head_bp_.mutable_cpu_data();
  Dtype *weight = weight_.mutable_cpu_data(), *weight_bp = weight_bp_.mutable_cpu_data(); 

  caffe_set(superpixel_num_, int(-1), head);
  caffe_set(h * w, int(-1), head_bp);

  vector<float> pixel_tot(superpixel_num_, 0.);
  for (int i=0; i<sh * sw; i++)
    pixel_tot[bottom_sp[i]]++;

  int edge_tot = 0;
  map<int, float> hash;
  for (int i=0; i<h; i++)
    for (int j=0; j<w; j++)
    {
      int index = i * w + j;
      const int stx = i*factor_ - pad, sty = j*factor_ - pad;
      hash.clear();
      for (int dx = 0; dx < kernel_size; dx++)
        if (dx + stx >=0 && dx + stx < sh)
          for (int dy = 0; dy < kernel_size; dy++)
            if (dy + sty >=0 && dy + sty < sw)
            {
              const int sindex = (dx + stx) * sw + (dy + sty);
              if (hash.find(bottom_sp[sindex]) == hash.end())
                hash[bottom_sp[sindex]] = float(filter.data_at(0, 0, dx, dy));
              else
                hash[bottom_sp[sindex]] += float(filter.data_at(0, 0, dx, dy));
            }

      for (map<int,float>::iterator iter=hash.begin(); iter!=hash.end(); iter++)
      {
        CHECK(edge_tot < ending_bp_.count());
        CHECK(index < head_bp_.count());
        CHECK(iter->first < head_.count());

        ending_bp[edge_tot] = iter->first;
        weight_bp[edge_tot] = Dtype(iter->second / pixel_tot[iter->first]) ;
        prev_bp[edge_tot] = head_bp[index];
        head_bp[index] = edge_tot;

        ending[edge_tot] = index;
        weight[edge_tot] = Dtype(iter->second / pixel_tot[iter->first]);
        prev[edge_tot] = head[iter->first];
        head[iter->first] = edge_tot;
        edge_tot++;
      }
    }
  caffe_gpu_set(count, Dtype(0.), top_data);
  AveSuperpixelPoolForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, bottom_data, bottom[0]->channels(), h, w, superpixel_num_, head_.gpu_data(), ending_.gpu_data(), weight_.gpu_data(), prev_.gpu_data(), top_data);
  
  CUDA_POST_KERNEL_CHECK;
}

template <typename Dtype>
__global__ void AveSuperpixelPoolBackward(const int nthreads,
    const Dtype* const top_diff, const int channels,
    const int height, const int width, const int superpixel_num_,
    const int* head, const int* ending, const Dtype* weight, const int* prev,
    Dtype* const bottom_diff) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int w = index % width;
    const int h = (index / width) % height;
    const int c = (index / width / height) % channels;
    const int base_index = c * superpixel_num_;
    for (int h_t=head[h * width + w]; h_t!=-1; h_t = prev[h_t])
      bottom_diff[index] += top_diff[base_index + ending[h_t]] * weight[h_t];
  }
}

template <typename Dtype>
void SuperpixelPoolingLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  if (!propagate_down[0]) {
    return;
  }
  const Dtype* top_diff = top[0]->gpu_diff();
  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
  int count = bottom[0]->count();
  int h = bottom[0]->height(), w = bottom[0]->width();

  caffe_gpu_set(count, Dtype(0.), bottom_diff);

  AveSuperpixelPoolBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
     count, top_diff, bottom[0]->channels(), h, w, superpixel_num_, head_bp_.gpu_data(), ending_bp_.gpu_data(), weight_bp_.gpu_data(), prev_bp_.gpu_data(), bottom_diff);

  CUDA_POST_KERNEL_CHECK;
}


INSTANTIATE_LAYER_GPU_FUNCS(SuperpixelPoolingLayer);


}  // namespace caffe
