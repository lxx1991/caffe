#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layers/learnt_mapping_layer.hpp"
#include "caffe/util/math_functions.hpp"



namespace caffe {


template <typename Dtype>
__global__ void LearntMappingForward(const int nthreads,
          const Dtype* bottom_data, const int num, const int channels,
		  const int height, const int width, const Dtype* filter, const int filter_channels, const int filter_height,
    	  const int filter_width, Dtype* top_data) 
{
	  CUDA_KERNEL_LOOP(index, nthreads) {
	  		int w = index % width;
	  		int h = (index / width) % height;
	  		int c = (index / width / height) % channels;
	  		int n = index / width / height / channels;
	  		int filter_th = h * width + w;
	  		for (int i=0; i<filter_height; i++)
	  			for (int j=0; j<filter_width; j++)
	  					top_data[index] += bottom_data[((n * channels  + c) * filter_height + i)*filter_width + j] * filter[((n * filter_channels + filter_th) * filter_height + i)*filter_width + j];
	  }
}


template <typename Dtype>
__global__ void LearntMappingBackward(const int nthreads,
          Dtype* bottom_diff, const Dtype* bottom_data, const int num, const int channels,
		  const int height, const int width, Dtype* filter_diff, const Dtype* filter, const int filter_channels, const int filter_height,
    	  const int filter_width, const Dtype* top_diff) 
{
	  CUDA_KERNEL_LOOP(index, nthreads) {
	  		int w = index % width;
	  		int h = (index / width) % height;
	  		int c = (index / width / height) % channels;
	  		int n = index / width / height / channels;
	  		int filter_th = h * width + w;
	  		for (int i=0; i<filter_height; i++)
				for (int j=0; j<filter_width; j++)
				{
					bottom_diff[((n * channels  + c) * filter_height + i)*filter_width + j] += top_diff[index] * filter[((n * filter_channels + filter_th) * filter_height + i)*filter_width + j];
					filter_diff[((n * filter_channels + filter_th) * filter_height + i)*filter_width + j] += top_diff[index] * bottom_data[((n * channels  + c) * filter_height + i)*filter_width + j];
				}
	  }
}


template <typename Dtype>
void LearntMappingLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {

  const Dtype* bottom_data = bottom[0]->gpu_data();
  const Dtype* bottom_filter = bottom[1]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  int count = top[0]->count();
  caffe_gpu_set(count, Dtype(0.), top_data);

  LearntMappingForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, bottom_data, top[0]->num(), top[0]->channels(), top[0]->height(), top[0]->width(), bottom_filter, bottom[1]->channels(), bottom[1]->height(), bottom[1]->width(), top_data);
  
  CUDA_POST_KERNEL_CHECK;
}



template <typename Dtype>
void LearntMappingLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {

  int count = top[0]->count();
  caffe_gpu_set(bottom[0]->count(), Dtype(0.), bottom[0]->mutable_gpu_diff());
  caffe_gpu_set(bottom[1]->count(), Dtype(0.), bottom[1]->mutable_gpu_diff());

  int filter_height = int(std::sqrt(bottom[1]->channels()));
  int filter_width = filter_height;
  CHECK_EQ(filter_height * filter_width, bottom[1]->channels());

  LearntMappingBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, bottom[0]->mutable_gpu_diff(), bottom[0]->gpu_data(), top[0]->num(), top[0]->channels(), top[0]->height(), top[0]->width(), bottom[1]->mutable_gpu_diff(), bottom[1]->gpu_data(), bottom[1]->channels(), bottom[1]->height(), bottom[1]->width(), top[0]->gpu_diff());
  
  CUDA_POST_KERNEL_CHECK;
}


INSTANTIATE_LAYER_GPU_FUNCS(LearntMappingLayer);


}  // namespace caffe